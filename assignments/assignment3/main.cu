#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix.h"
//#define DEBUG

int main()
{
    int nArray[] = {100, 200, 500, 1000, 2000, 5000, 10000};
    int indexLength = sizeof(nArray)/sizeof(int);
    for (int i=0; i<indexLength; i++)
    {
        struct timeval start, end;
        gettimeofday(&start, NULL);

        int n = nArray[i];
        double **a =getmatA(n);
        double **b =getmatB(n);
        double **c =allocate(n);
        double *dev_a, *dev_b, *dev_c;
        int *dev_n;
        int size = sizeof(double)*n*n;
        int size2 = sizeof(double)*n;
        hipMalloc( (void**)&dev_b, size );
        hipMemcpy( dev_b, &b[0][0], size, hipMemcpyHostToDevice );
        hipMalloc( (void**)&dev_a, size2 );
        hipMalloc( (void**)&dev_c, size2 );
        hipMalloc( (void**)&dev_n,sizeof(int) );
        hipMemcpy( dev_n, &n, sizeof(int), hipMemcpyHostToDevice );

        for(int i=0;i<n;i++)
        {
#ifdef DEBUG
            if(i%100 ==0)
            {
                gettimeofday(&end, NULL);
                double delta = ((end.tv_sec  - start.tv_sec) * 1000000u +
                        end.tv_usec - start.tv_usec) / 1.e6;
                printf("%d,%f\n", i,delta);
            }
#endif
            hipMemcpy( dev_a, &a[i][0], size2, hipMemcpyHostToDevice );
            matrixMultiplication<<< n/100, 100 >>>( dev_a, dev_b, dev_c, dev_n );
            hipMemcpy( &c[i][0], dev_c, size2, hipMemcpyDeviceToHost );
        }
        hipFree( dev_a );
        hipFree( dev_b );
        hipFree( dev_c );
        hipFree( dev_n );

        gettimeofday(&end, NULL);
        double delta = ((end.tv_sec  - start.tv_sec) * 1000000u +
                end.tv_usec - start.tv_usec) / 1.e6;
        printf("%d,%f\n", n,delta);
#ifdef DEBUG
        printMat(a,n);
        printMat(b,n);
        printMat(c,n);
#endif
        free(&a[0][0]);
        free(&b[0][0]);
        free(&c[0][0]);

    }
return 0;
}
